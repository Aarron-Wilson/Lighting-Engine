#include "hip/hip_runtime.h"
﻿#define GLEW_STATIC
#include <GL/glew.h>
#include <GLFW/glfw3.h>
#include "hip/hip_runtime.h"
#include ""
#include "cuda_gl_interop.h"


#include <thread>
#include <chrono>
#include <stdio.h>
#include <iostream>
#include "Light.h"
#include <vector>

const int LIGHT_SAMPLE_POINTS = 10;
const float LIGHT_SAMPLE_POINT_RADIUS = 15.0;
const int LIGHT_COUNT = 2;
const int OBSTACLE_COUNT = 2;

__device__
typedef struct light
{
    float lightx;
    float lighty;
    float startingIntensity;
    float radius;
    //float visibilityLeftRotations[10];
    //float visibilityRightRotations[10];
    //float visibilityRadius[10];

    //float testxpoints[LIGHT_SAMPLE_POINTS];
   // float testypoints[LIGHT_SAMPLE_POINTS];
};

__device__
typedef struct obstacle
{
    int x;
    int y;
    int width;
    int height;
};

//float* getMapData();
void errorCheck(int errorVal);
void endProgram();
void processKey(GLFWwindow* window);

//const int MAP_WIDTH = 640;
//const int MAP_HEIGHT = 480;

const int MAP_WIDTH = 1500;
const int MAP_HEIGHT = 700;

GLFWwindow* window;

GLuint lightMapTexture;
hipGraphicsResource* lightMapTextureResource;
uchar4* g_dstBuffer = NULL;
size_t g_BufferSize = 0;

light* h_Lights;
int lightIndex = 0;
light* d_Lights;
obstacle* h_obstacles;
int obstacleIndex = 0;
obstacle* d_obstacles;

__device__
float amax(float a, float b) {
    if (a >= b) {
        return a;
    }

    return b;
}

__device__
float amin(float a, float b) {
    if (a <= b) {
        return a;
    }

    return b;
}


__device__
float distance(float x1, float y1, float x2, float y2) {
    float xdif = x1 - x2;
    float ydif = y1 - y2;
    //return std::sqrt((xdif * xdif) + (ydif * ydif));
    return sqrt((xdif * xdif) + (ydif * ydif));
}

__device__
float rotation(float x1, float y1, float x2, float y2) {
    float xdif, ydif;

    xdif = x2 - x1;
    ydif = y2 - y1;
    

    float rot = atan(abs(ydif) / abs(xdif));

    if (xdif >= 0.0) {
        if (ydif >= 0.0) {
            return rot;
        }
        else
        {
            return (2 * 3.14) - rot;
        }
    }
    else
    {
        if (ydif >= 0.0) {
            return 3.14 - rot;
        }
        else
        {
            return 3.14 + rot;
        }
    }

}

__device__
bool pointOnLine(float l1x, float l1y, float l2x, float l2y, float px, float py)
{
    float leftx, rightx,lefty,righty;
    leftx = amin(l1x, l2x);
    rightx = amax(l1x, l2x);

    if (!((px >= leftx) && (px <= rightx))) {
        return false;
    }

    if (l1x <= l2x) {
        lefty = l1y;
        righty = l2y;
    }
    else
    {
        lefty = l2y;
        righty = l1y;
    }

    float pct = (px - leftx) / (rightx - leftx);
    float targetY = ((1 - pct) * lefty) + (pct * righty);

    return abs(py - targetY) < 4.0;
}

__device__
inline bool withinBox(float bx, float by, float bw, float bh, float tx, float ty) {
    return ((tx > bx) && (tx < (bx + bw)) && (ty > by) && (ty < (by + bh)));
}

__device__
bool clearPath(int targetx, int targety, light tlight, obstacle* obstacleList, int obstacleCount)
{
    float SAMPLE_POINT_RADIUS = 15.0;
    float leftx, rightx, topy, bottomy;
    leftx = 99999.0;
    rightx = -999999.0;
    topy = 9999999;
    bottomy = -99999.0;
    leftx = amin(leftx, tlight.lightx - SAMPLE_POINT_RADIUS);
    leftx = amin(leftx, targetx);
    rightx = amax(rightx, tlight.lightx + SAMPLE_POINT_RADIUS);
    rightx = amax(rightx, targetx);
    topy = amin(topy, tlight.lighty - SAMPLE_POINT_RADIUS);
    topy = amin(topy, targety);
    bottomy = amax(bottomy, tlight.lighty + SAMPLE_POINT_RADIUS);
    bottomy = amax(bottomy, targety);

    for (int obstacleIndex = 0; obstacleIndex < obstacleCount; obstacleIndex++) {
        if (withinBox(leftx, topy, rightx - leftx, bottomy - topy, obstacleList[obstacleIndex].x, obstacleList[obstacleIndex].y)) {
            return false;
        }

        if (withinBox(leftx, topy, rightx - leftx, bottomy - topy, obstacleList[obstacleIndex].x + obstacleList[obstacleIndex].width, obstacleList[obstacleIndex].y)) {
            return false;
        }

        if (withinBox(leftx, topy, rightx - leftx, bottomy - topy, obstacleList[obstacleIndex].x + obstacleList[obstacleIndex].width, obstacleList[obstacleIndex].y + obstacleList[obstacleIndex].height)) {
            return false;
        }

        if (withinBox(leftx, topy, rightx - leftx, bottomy - topy, obstacleList[obstacleIndex].x, obstacleList[obstacleIndex].y + obstacleList[obstacleIndex].height)) {
            return false;
        }
    }

    return true;
}

__device__
float getLightValue(float startingIntensity, int lightx, int lighty,float lightRadius,int targetx, int targety) {
    float xdif, ydif;
    xdif = lightx - targetx;
    ydif = lighty - targety;
    float distance = sqrt((xdif * xdif) + (ydif * ydif));
    float pct = distance / lightRadius;
    pct *= 4.0;
    float multiplier = 1.0 / (1.0 + pct + (pct * pct));
    multiplier -= 0.048;
    
    if (distance > lightRadius) {
        return 0.0;
    }

    if (multiplier < 0.0) {
        return 0.0;
    }

    return startingIntensity * multiplier;
    //return startingIntensity - (startingIntensity * (distance / lightRadius));
}

__device__
bool rayTrace(float x1, float y1, float x2, float y2, obstacle* obstacleList, int obstacleCount) {
    float pointsPerDistance = 2.0;
    int points = (int) (distance(x1, y1, x2, y2) * pointsPerDistance);
    float xstep, ystep;
    float cx, cy;

    xstep = (x2 - x1) / ((float)points);
    ystep = (y2 - y1) / ((float)points);

    cx = x1;
    cy = y1;

    for (int i = 0; i < points; i++) {
        for (int obstacleIndex = 0; obstacleIndex < obstacleCount; obstacleIndex++) {
            if (withinBox(obstacleList[obstacleIndex].x, obstacleList[obstacleIndex].y, obstacleList[obstacleIndex].width, obstacleList[obstacleIndex].height, cx, cy)) {
                return false;
            }

            cx += xstep;
            cy += ystep;
        }
    }

    return true;
}

__device__
float fullLight(int targetx, int targety, light tlight, obstacle* obstacleList, int obstacleCount) {
    float tx = (float) targetx;
    float ty = (float) targety;

    float pointToLightRotation = rotation( tlight.lightx, tlight.lighty,tx,ty);
    float firstRotation, secondRotation;

    firstRotation = pointToLightRotation + (3.14 * 0.5);
    
    if (firstRotation > (2 * 3.14)) {
        firstRotation -= (2 * 3.14);
    }

    secondRotation = firstRotation + 3.14;

    if (secondRotation > (2 * 3.14)) {
        secondRotation -= 2 * 3.14;
    }

    float xpoints[2];
    float ypoints[2];

    xpoints[0] = tlight.lightx + (15.0 * cos(firstRotation));
    ypoints[0] = tlight.lighty + (15.0 * sin(firstRotation));

    xpoints[1] = tx + (15.0 * cos(secondRotation));
    ypoints[1] = ty + (15.0 * sin(secondRotation));

    if (!(rayTrace(xpoints[0], ypoints[0], tx, ty, obstacleList, obstacleCount))) {
        return false;
    }

    if (!(rayTrace(xpoints[1], ypoints[1], tx, ty, obstacleList, obstacleCount))) {
        return false;
    }

    return true;
}

__device__
float getLightAmount(int targetx, int targety, light tlight,obstacle* obstacleList, int obstacleCount) {
    int lightSamplePoints = 10;

    light thelight = tlight;
    
    const float SAMPLE_POINTS_PER_DISTANCE = 0.7;
    
    float originalIntensity = getLightValue(thelight.startingIntensity, thelight.lightx, thelight.lighty, thelight.radius, targetx, targety);
    float intensityPerConnection = originalIntensity / (float)lightSamplePoints;
    
    if ((intensityPerConnection) < 0.00001) {
        return (float) 0.0;
    }

    if (clearPath(targetx, targety, tlight, obstacleList, obstacleCount)) {
        return originalIntensity;
    }

    //if (fullLight(targetx, targety, tlight, obstacleList, obstacleCount)) {
    //    return originalIntensity;
    //}

    


    float currentx, currenty,xinc,yinc;
    currentx = currenty = xinc = yinc = 0.0;
    //return 0.5;
    int samplePoints = (int) (distance(targetx, targety, thelight.lightx, thelight.lighty) * SAMPLE_POINTS_PER_DISTANCE);
    float result = 0.0;

    bool hitObstacle = false;

    //return 0.5;

    for (int i = 0; i < lightSamplePoints; i++) {
        currentx = targetx;
        currenty = targety;

        float rotation = ((float) i / (float) lightSamplePoints) * 2.0 * 3.14;
        
        float testXPoint = thelight.lightx + 15.0 * cos(rotation);
        float testYPoint = thelight.lighty + 15.0 * sin(rotation);

        xinc = (testXPoint - targetx) / (float) samplePoints;
        yinc = (testYPoint - targety) / (float) samplePoints;
        float intensityAtTestPoint = getLightValue(thelight.startingIntensity, thelight.lightx, thelight.lighty, thelight.radius, testXPoint, testYPoint);
        //float finalIntensity = getLightValue(intensityAtTestPoint, thelight.testxpoints[i], thelight.testypoints[i], thelight.radius, targetx, targety);
        //finalIntensity /= (float) lightSamplePoints;

        hitObstacle = false;

        for (int testPoint = 0;testPoint < samplePoints;testPoint++) {
            for (int obstacleIndex = 0; obstacleIndex < obstacleCount; obstacleIndex++) {
                if (withinBox(obstacleList[obstacleIndex].x, obstacleList[obstacleIndex].y, obstacleList[obstacleIndex].width, obstacleList[obstacleIndex].height, currentx, currenty))
                {
                    hitObstacle = true;
                    break;
                }
            }

            if (hitObstacle) {
                break;
            }

            currentx += xinc;
            currenty += yinc;
        }

        if (!(hitObstacle)) {
            result += intensityPerConnection;
            //result += finalIntensity;
        }
    }

    return result;
}

__global__ void calculations(float* mapdata,light* firstLight,obstacle* obstacles)
{   
    int lightCount = 1;// (*lcount);
    int obstacleCount = 1;//(*oCount);
    //int i = threadIdx.x;
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    
    int y = (int)(i / 640);
    int x = i - (y * 640);

    if (i < 0 || i >(640 * 480)) {
        return;
    }

    float startingIntensity = 0.0;
    //printf("%d", sizeof(firstLight));
    //printf("%d", sizeof(*firstLight));
    
    for (int lightIndex = 0; lightIndex < lightCount; lightIndex++) {
        startingIntensity = firstLight[lightIndex].startingIntensity;
        float lightX = firstLight[lightIndex].lightx;//100.0;
        float lightY = firstLight[lightIndex].lighty;//100.0;
        float radius = firstLight[lightIndex].radius;//300;
        //mapdata[i] -= getLightValue(startingIntensity, lightX, lightY, radius, x, y);
        mapdata[i] -= getLightAmount(x, y, firstLight[lightIndex], obstacles, obstacleCount);
        
        /*
        for (int testIndex = 0; testIndex < 50; testIndex++)
        {
            int index = firstLight[lightIndex].testxpoints[testIndex] + (firstLight[lightIndex].testypoints[testIndex] * 640);
            if ((index >= 0) && (index < (460 * 640))) {
                mapdata[index] = 0.0;
            }
        }
        */
    }

    __syncthreads();
}

__global__ void calc(float* output) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;

    int y = (int)(i / 640);
    int x = i - (y * 640);

    if (i < 0 || i >(640 * 480)) {
        return;
    }

    output[i * 4] = 34.0;
}

__global__ void forReal(uchar4* dst,light* lights,obstacle* obstacles) {
    const int OBSTACLE_COUNT = 2;

    //int x = blockIdx.x * blockDim.x + threadIdx.x;
    //int y = blockIdx.y * blockDim.y + threadIdx.y;

    //int i = threadIdx.x + blockIdx.x * blockDim.x;

    int i = ((blockDim.x * blockDim.y) * blockIdx.x) + (threadIdx.y * blockDim.x) + threadIdx.x;
    int y = (int) (i / MAP_WIDTH);
    int x = i - (y * MAP_WIDTH);

    if (i < 0 || i > (MAP_WIDTH * MAP_HEIGHT)) {
        return;
    }

    if (x > MAP_WIDTH || y > MAP_HEIGHT) {
        return;
    }

    /*
    float randomPoint1x, randomPoint1y, randomPoint2x, randomPoint2y;
    randomPoint1x = 350.0;
    randomPoint1y = 350.0;
    randomPoint2x = 400.0;
    randomPoint2y = 370.0;
    
    float randomPointRotation = rotation(randomPoint1x, randomPoint1y, randomPoint2x, randomPoint2y);

    if (distance(randomPoint1x, randomPoint1y, x, y) < 5.0) {
        dst[i] = make_uchar4(0.0, 100.0, 0.0, 255.0);
        return;
    }

    if (distance(randomPoint2x, randomPoint2y, x, y) < 5.0) {
        dst[i] = make_uchar4(0.0, 100.0, 0.0, 255.0);
        return;
    }

    if (pointOnLine(randomPoint1x, randomPoint1y, randomPoint2x, randomPoint2y, x, y)) {
        dst[i] = make_uchar4(0.0, 0.0, 100.0, 255.0);
        return;
    }


    float perpPoint1x, perpPoint1y, perpPoint2x, perpPoint2y;
    float perpRotation1, perpRotation2;
    perpRotation1 = randomPointRotation + (0.5 * 3.14);

    if (perpRotation1 > (2 * 3.14)) {
        perpRotation1 -= 2 * 3.14;
    }

    perpRotation2 = perpRotation1 + 3.14;
    if (perpRotation2 > (2 * 3.14)) {
        perpRotation2 -= 2 * 3.14;
    }

    perpPoint1x = randomPoint1x + (30.0 * cos(perpRotation1));
    perpPoint1y = randomPoint1y + (30.0 * sin(perpRotation1));

    if (pointOnLine(randomPoint1x, randomPoint1y, perpPoint1x, perpPoint1y, x, y)) {
        dst[i] = make_uchar4(100.0, 0.0, 100.0, 255.0);
        return;
    }

    perpPoint2x = randomPoint1x + (30.0 * cos(perpRotation2));
    perpPoint2y = randomPoint1y + (30.0 * sin(perpRotation2));

    if (pointOnLine(randomPoint1x, randomPoint1y, perpPoint2x, perpPoint2y, x, y)) {
        dst[i] = make_uchar4(100.0, 0.0, 100.0, 255.0);
        return;
    }


    float lightx, lighty, point1x, point1y, point2x, point2y;
    float testRotation,leftRotation, rightRotation,pct;
    lightx = 150.0;
    lighty = 150.0;
    point1x = 330.9;
    point1y = 330.0;
    point2x = 250.0;
    point2y = 200.0;

    rightRotation = rotation(lightx, lighty, point1x, point1y);
    leftRotation = rotation(lightx, lighty, point2x, point2y);
    testRotation = rotation(lightx, lighty, x, y);
    pct = (testRotation - leftRotation) / (rightRotation - leftRotation);

    float boundaryx, boundaryy,finalDistance;
    boundaryx = ((pct * point1x) + ((1 - pct) * point2x));
    boundaryy = ((pct * point1y) + ((1 - pct) * point2y));
    finalDistance = distance(lightx, lighty, boundaryx, boundaryy);
    //finalDistance = distance(lightx, lighty, point1x, point1y);
    if (testRotation >= leftRotation && testRotation <= rightRotation && distance(lightx,lighty,x,y) < finalDistance) {
        dst[i] = make_uchar4(0.0, 100.0,0.0, 255.0);
        return;
    }


    if (distance(x, y, lightx, lighty) < 3.0) {
        dst[i] = make_uchar4(0.0, 0.0, 100.0, 255.0);
        return;
    }

    if (distance(x, y, point1x, point1y) < 3.0) {
        dst[i] = make_uchar4(0.0, 0.0, 100.0, 255.0);
        return;
    }

    if (distance(x, y, point2x, point2y) < 3.0) {
        dst[i] = make_uchar4(0.0, 0.0, 100.0, 255.0);
        return;
    }


    float tx, ty,radius,firstRotation,secondRotation;
    float rot,dis;
    tx = 100.0;
    ty = 100.0;
    firstRotation = 0.0 * 3.14;
    secondRotation = 0.003 * 3.14;
    dis = distance(tx, ty, x, y);
    ///rotation = atan((y - ty) / (x - tx));
    rot = rotation(tx, ty, x, y);

    if (rot >= firstRotation && rot <= secondRotation) {
        //dst[i] = make_uchar4(0.0, 100.0, 100.0, 255.0);
        //return;
    }
    */

    float Red = 0.0;
    float Green = 0.7 * 255.0;
    float Blue = 0.7 * 255.0;
    float Alpha = 0.91;
    float startingIntensity = 0.0;

    bool withinObstacle = false;

    for (int ob = 0; ob < OBSTACLE_COUNT; ob++) {
        if (withinBox(obstacles[ob].x, obstacles[ob].y, obstacles[ob].width, obstacles[ob].height, x, y)) {
            withinObstacle = true;
            break;
        }
    }
    if (!(withinObstacle)) {
        for (int lightIndex = 0; lightIndex < 100; lightIndex++) {
            if (lights[lightIndex].lightx == -1) {
                break;
            }

            startingIntensity = lights[lightIndex].startingIntensity;
            float lightX = lights[lightIndex].lightx;//100.0;
            float lightY = lights[lightIndex].lighty;//100.0;
            float radius = lights[lightIndex].radius;//300;
            //mapdata[i] -= getLightValue(startingIntensity, lightX, lightY, radius, x, y);
            Alpha -= getLightAmount(x, y, lights[lightIndex], obstacles, OBSTACLE_COUNT);
        }
    }

    

    if (Alpha > 1.0) {
        Alpha = 1.0;
    }

    if (Alpha < 0.0) {
        Alpha = 0.0;
    }
    
    Red -= Red * Alpha;
    Blue -= Blue * Alpha;
    Green -= Green * Alpha;
    
    dst[i] = make_uchar4(Red, Green, Blue, 255.0);
    
    

    //__syncthreads();
}

__global__ void calculateVisibilityTriangles(light* lights, obstacle* obstacles) {

}

void DeleteTexture(GLuint& texture)
{
    if (texture != 0)
    {
        glDeleteTextures(1, &texture);
        texture = 0;
    }
}

void CreateTexture(GLuint& texture, unsigned int width, unsigned int height)
{
    // Make sure we don't already have a texture defined here
    DeleteTexture(texture);

    glGenTextures(1, &texture);
    glBindTexture(GL_TEXTURE_2D, texture);

    // set basic parameters
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_S, GL_CLAMP_TO_EDGE);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_T, GL_CLAMP_TO_EDGE);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_NEAREST);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_NEAREST);

    // Create texture data (4-component unsigned byte)
    glTexImage2D(GL_TEXTURE_2D, 0, GL_RGBA, width, height, 0, GL_RGBA, GL_UNSIGNED_BYTE, NULL);

    // Unbind the texture
    glBindTexture(GL_TEXTURE_2D, 0);
}

void DisplayImage(GLuint texture, unsigned int x, unsigned int y, unsigned int width, unsigned int height)
{
    glBindTexture(GL_TEXTURE_2D, texture);
    glEnable(GL_TEXTURE_2D);
    glDisable(GL_DEPTH_TEST);
    glDisable(GL_LIGHTING);
    glTexEnvf(GL_TEXTURE_ENV, GL_TEXTURE_ENV_MODE, GL_REPLACE);

    glMatrixMode(GL_PROJECTION);
    glPushMatrix();
    glLoadIdentity();
    glOrtho(-1.0, 1.0, -1.0, 1.0, -1.0, 1.0);

    glMatrixMode(GL_MODELVIEW);
    glLoadIdentity();

    glPushAttrib(GL_VIEWPORT_BIT);
    glViewport(x, y, width, height);

    glBegin(GL_QUADS);
    glTexCoord2f(0.0, 0.0); glVertex3f(-1.0, -1.0, 0.5);
    glTexCoord2f(1.0, 0.0); glVertex3f(1.0, -1.0, 0.5);
    glTexCoord2f(1.0, 1.0); glVertex3f(1.0, 1.0, 0.5);
    glTexCoord2f(0.0, 1.0); glVertex3f(-1.0, 1.0, 0.5);
    glEnd();

    glPopAttrib();

    glMatrixMode(GL_PROJECTION);
    glPopMatrix();

    //glDisable(GL_TEXTURE_2D);
}

void drawImage(GLuint file,float x,float y,float w, float h)
{
    //glColor3f(0.0, 100.7, 255.0);
    //glVertex3f(x, y, 0.0f);
    //glVertex3f(x, y + h, 0.0f);
    //glVertex3f(x + w, y + h, 0.0f);
    //glVertex3f(x + w, y, 0.0f);
    glMatrixMode(GL_MODELVIEW);
    glLoadIdentity();

    glPushAttrib(GL_VIEWPORT_BIT);
    glViewport(x, y, MAP_WIDTH, MAP_HEIGHT);

    glTexEnvf(GL_TEXTURE_ENV, GL_TEXTURE_ENV_MODE, GL_DECAL);


    glPushMatrix();
    glTranslatef(x, y, 0.0);

    glBindTexture(GL_TEXTURE_2D, file);
    glEnable(GL_TEXTURE_2D);

    glBegin(GL_QUADS);
    glTexCoord2f(0.0, 0.0); glVertex3f(x, y, 0.0f);
    glTexCoord2f(0.0, 1.0); glVertex3f(x, y + h, 0.0f);
    glTexCoord2f(1.0, 1.0); glVertex3f(x + w, y + h, 0.0f);
    glTexCoord2f(1.0, 0.0); glVertex3f(x + w, y, 0.0f);
    glEnd();

    glPopMatrix();
}

void initializeLightsAndObstacles() {
    //Standard
    h_Lights = (light*)malloc(sizeof(light) * LIGHT_COUNT);

    //Unified memory
    //hipMallocManaged(&h_Lights, LIGHT_COUNT * sizeof(light));

    //Mapped
    //errorCheck(hipHostAlloc(&h_Lights, sizeof(light) * LIGHT_COUNT, hipHostMallocMapped));
    //errorCheck(hipHostGetDevicePointer(&d_Lights, h_Lights, 0));

    for (int i = 0; i < LIGHT_COUNT; i++) {
        h_Lights[i].lightx = -1;
        h_Lights[i].lighty = -1;
        h_Lights[i].radius = -1;
        h_Lights[i].startingIntensity = -1;
    }

    h_Lights[0].lightx = 100;
    h_Lights[0].lighty = 100;
    h_Lights[0].radius = 300;
    h_Lights[0].startingIntensity = 0.8;

    h_Lights[1].lightx = 150;
    h_Lights[1].lighty = 300;
    h_Lights[1].radius = 300;
    h_Lights[1].startingIntensity = 0.8;

    //for (int randomIndex = 0; randomIndex < LIGHT_SAMPLE_POINTS; randomIndex++) {
    //    float rotation = ((float)randomIndex / (float)LIGHT_SAMPLE_POINTS) * 2.0 * 3.14;
    //    h_Lights[0].testxpoints[randomIndex] = h_Lights[0].lightx + LIGHT_SAMPLE_POINT_RADIUS * cos(rotation);
    //    h_Lights[0].testypoints[randomIndex] = h_Lights[0].lighty + LIGHT_SAMPLE_POINT_RADIUS * sin(rotation);
    //}

    h_obstacles = (obstacle*)malloc(sizeof(obstacle) * OBSTACLE_COUNT);

    h_obstacles[0].x = 200;
    h_obstacles[0].y = 200;
    h_obstacles[0].width = 25;
    h_obstacles[0].height = 25;

    h_obstacles[1].x = 325;
    h_obstacles[1].x = 325;
    h_obstacles[1].y = 200;
    h_obstacles[1].width = 25;
    h_obstacles[1].height = 25;

    errorCheck(hipMalloc((light**)&d_Lights, sizeof(light) * LIGHT_COUNT));
    errorCheck(hipHostMalloc((void**)&d_Lights, sizeof(light) * LIGHT_COUNT));
    errorCheck(hipMemcpy(d_Lights, h_Lights, sizeof(light) * LIGHT_COUNT, hipMemcpyHostToDevice));

    errorCheck(hipMalloc((obstacle**)&d_obstacles, sizeof(obstacle) * OBSTACLE_COUNT));
    errorCheck(hipMemcpy(d_obstacles, h_obstacles, sizeof(obstacle) * OBSTACLE_COUNT, hipMemcpyHostToDevice));
}

void initializeOpenGL()
{
    /* Initialize the library */
    if (!glfwInit())
        return;

    /* Create a windowed mode window and its OpenGL context */
    window = glfwCreateWindow(MAP_WIDTH, MAP_HEIGHT, "Light Sim", NULL, NULL);
    if (!window)
    {
        glfwTerminate();
        return;
    }

    /* Make the window's context current */
    glfwMakeContextCurrent(window);
    glEnable(GL_BLEND);
    glBlendFunc(GL_SRC_ALPHA, GL_ONE_MINUS_SRC_ALPHA);
    //glfwGetFramebufferSize(window, MAP_WIDTH, MAP_HEIGHT);
    glMatrixMode(GL_PROJECTION);
    glLoadIdentity();
    //glOrtho(0, MAP_WIDTH, MAP_HEIGHT, 0, -1, 1);
    glOrtho(0, MAP_WIDTH, MAP_HEIGHT, 0, 0, 1);

    glewInit();
}

float distanceH(float x1, float y1, float x2, float y2) {
    float xdif = x1 - x2;
    float ydif = y1 - y2;
    //return std::sqrt((xdif * xdif) + (ydif * ydif));
    return sqrt((xdif * xdif) + (ydif * ydif));
}

inline bool withinBoxH(float bx, float by, float bw, float bh, float tx, float ty) {
    return ((tx > bx) && (tx < (bx + bw)) && (ty > by) && (ty < (by + bh)));
}

bool rayTraceH(float x1, float y1, float x2, float y2, obstacle* obstacleList, int obstacleCount) {
    float pointsPerDistance = 0.3;
    int points = (int)(distanceH(x1, y1, x2, y2) * pointsPerDistance);
    float xstep, ystep;
    float cx, cy;

    xstep = (x2 - x1) / ((float)points);
    ystep = (y2 - y1) / ((float)points);

    cx = x1;
    cy = y1;

    for (int i = 0; i < points; i++) {
        for (int obstacleIndex = 0; obstacleIndex < obstacleCount; obstacleIndex++) {
            if (withinBoxH(obstacleList[obstacleIndex].x, obstacleList[obstacleIndex].y, obstacleList[obstacleIndex].width, obstacleList[obstacleIndex].height, cx, cy)) {
                return false;
            }

            cx += xstep;
            cy += ystep;
        }
    }

    return true;
}



float rotationH(float x1, float y1, float x2, float y2) {
    float xdif, ydif;
    xdif = x2 - x1;
    ydif = y2 - y1;

    float rot = atan(abs(ydif) / abs(xdif));

    if (xdif >= 0.0) {
        if (ydif >= 0.0) {
            return rot;
        }
        else {
            return (2 * 3.14) - rot;
        }
    }
    else {
        if (ydif >= 0.0) {
            return 3.14 - rot;
        }
        else {
            return 3.14 + rot;
        }
    }
}



void calculateVisibilityTrianglesAA(light* lights, obstacle* obstacles) {
    float lastLength = 0.0;
    float currentLength = 0.0;
    float currentRotation = 0.0;
    float rotationIncrement = (2 * 3.14) / 100.0;
    float perpRotation1, perpRotation2;
    float perp1x, perp1y, perp2x, perp2y;
    float traceX, traceY;
    float testX, testY;

    for (int lightIndex = 0; lightIndex < LIGHT_COUNT; lightIndex++) {
        lastLength = 0.0;
        currentLength = 0.0;
        currentRotation = 0.0;


        for (int i = -1; i < 100; i++) {
            perpRotation1 = currentRotation + (3.14 * 0.5);
            if (perpRotation1 > (2 * 3.14)) {
                perpRotation1 -= (2 * 3.14);
            }

            perpRotation2 = perpRotation1 + 3.14;
            if (perpRotation2 > (2 * 3.14)) {
                perpRotation2 -= (2 * 3.14);
            }

            perp1x = lights[lightIndex].lightx + (15.0 * cos(perpRotation1));
            perp1y = lights[lightIndex].lighty + (15.0 * sin(perpRotation1));
            perp2x = lights[lightIndex].lightx + (15.0 * cos(perpRotation2));
            perp2y = lights[lightIndex].lighty + (15.0 * sin(perpRotation2));

            traceX = 10.0 * cos(currentRotation);
            traceY = 10.0 * sin(currentRotation);

            testX = lights[lightIndex].lightx + traceX;
            testY = lights[lightIndex].lighty + traceY;

            currentLength = 0.0;
            while (currentLength < 280.0) {
                if (!(rayTraceH(perp1x, perp1y, testX, testY, obstacles, OBSTACLE_COUNT))) {
                    break;
                }

                if (!(rayTraceH(perp2x, perp2y, testX, testY, obstacles, OBSTACLE_COUNT))) {
                    break;
                }

                currentLength += 30.0;
                testX += traceX;
                testY += traceY;
            }

            if (!(i == -1)) {
                //lights[lightIndex].visibilityLeftRotations[i] = currentRotation - rotationIncrement;
                //lights[lightIndex].visibilityRightRotations[i] = currentRotation - rotationIncrement;

                if (lastLength <= currentLength)
                {
                    //lights[lightIndex].visibilityRadius[i] = lastLength;
                }
                else
                {
                    //lights[lightIndex].visibilityRadius[i] = currentLength;
                }
            }
            lastLength = currentLength;
        }
    }
}

int main()
{
    initializeOpenGL();
    initializeLightsAndObstacles();

    CreateTexture(lightMapTexture, MAP_WIDTH, MAP_HEIGHT);
    hipGraphicsGLRegisterImage(&lightMapTextureResource, lightMapTexture, GL_TEXTURE_2D, cudaGraphicsMapFlagsWriteDiscard);

    std::chrono::time_point<std::chrono::system_clock> lastFrameOutput,currentTime;
    std::chrono::duration<double>  elapsed;
    int frames = 0;

    lastFrameOutput = std::chrono::system_clock::now();

    /* Loop until the user closes the window */
    while (!glfwWindowShouldClose(window))
    {
        /* Render here */
        //glColor3f(0.0, 0.7, 0.7);
        //glClear(GL_COLOR_BUFFER_BIT);
        
        //glClearColor(0.0, 0.7, 0.7, 1.0);
        int window_width, window_height;
        
        glEnable(GL_BLEND);
        glBlendFunc(GL_SRC_ALPHA, GL_ONE_MINUS_SRC_ALPHA);
        //glfwGetFramebufferSize(window, MAP_WIDTH, MAP_HEIGHT);
        glMatrixMode(GL_PROJECTION);
        glLoadIdentity();
        //glOrtho(0, MAP_WIDTH, MAP_HEIGHT, 0, -1, 1);
        glOrtho(0, MAP_WIDTH, MAP_HEIGHT, 0, 0, 1);

        h_Lights[0].lightx += 5.0;
        h_Lights[1].lightx += 5.0;

        //calculateVisibilityTriangles(h_Lights, h_obstacles);
        // 
        //for (int randomIndex = 0; randomIndex < LIGHT_SAMPLE_POINTS; randomIndex++) {
         ///   float rotation = ((float)randomIndex / (float)LIGHT_SAMPLE_POINTS) * 2.0 * 3.14;
        //    h_Lights[0].testxpoints[randomIndex] = h_Lights[0].lightx + LIGHT_SAMPLE_POINT_RADIUS * cos(rotation);
        //    h_Lights[0].testypoints[randomIndex] = h_Lights[0].lighty + LIGHT_SAMPLE_POINT_RADIUS * sin(rotation);
        //}

        std::chrono::time_point<std::chrono::system_clock> copyStart, copyEnd;
        std::chrono::duration<double> copyTime;
        copyStart = std::chrono::system_clock::now();

        errorCheck(hipMemcpy(d_Lights, h_Lights, sizeof(light) * LIGHT_COUNT, hipMemcpyHostToDevice));

        copyEnd = std::chrono::system_clock::now();
        copyTime = copyEnd - copyStart;
        //std::cout << "Copy Time: " << copyTime.count() << "s\n";


        hipGraphicsResource_t resources[1] = { lightMapTextureResource };
        hipGraphicsMapResources(1, resources);
        hipArray* dstArray;
        hipGraphicsSubResourceGetMappedArray(&dstArray, lightMapTextureResource, 0, 0);
        //hipBindTextureToArray(texRef, srcArray);

        

        size_t bufferSize = MAP_WIDTH * MAP_HEIGHT * sizeof(uchar4);
        if (g_BufferSize != bufferSize)
        {
            if (g_dstBuffer != NULL)
            {
                hipFree(g_dstBuffer);
            }
            // Only re-allocate the global memory buffer if the screen size changes, 
            // or it has never been allocated before (g_BufferSize is still 0)
            g_BufferSize = bufferSize;
            hipMalloc(&g_dstBuffer, g_BufferSize);
        }

        

        std::chrono::time_point<std::chrono::system_clock> kernelstart,kernelend;
        std::chrono::duration<double> kernelTime;
        kernelstart = std::chrono::system_clock::now();


        dim3 threadsPerBlock(8,8);
        //dim3 numBlocks((MAP_WIDTH * MAP_HEIGHT) / threadsPerBlock.x, (MAP_WIDTH * MAP_HEIGHT) / threadsPerBlock.y);
        //MatAdd << <numBlocks, threadsPerBlock >> > (A, B, C);

        //calculateVisibilityTriangles << <1, 200 >> > (d_Lights, d_obstacles);


        //forReal << <615, 500 >> > (g_dstBuffer,d_Lights,d_obstacles);
        int numBlocks = (MAP_WIDTH * MAP_HEIGHT) / (8 * 8);
        numBlocks += 5;
        forReal << <numBlocks, threadsPerBlock >> > (g_dstBuffer, d_Lights, d_obstacles);
        //hipDeviceSynchronize();
        kernelend = std::chrono::system_clock::now();
        kernelTime = kernelend - kernelstart;
        //std::cout << "Kernel Time: " << kernelTime.count() << "s\n";

        

        hipMemcpyToArray(dstArray, 0, 0, g_dstBuffer, bufferSize, hipMemcpyDeviceToDevice);
        hipGraphicsUnmapResources(1, resources);
        drawImage(lightMapTexture, 0, 0, MAP_WIDTH, MAP_HEIGHT);

        

        //DisplayImage(lightMapTexture, 0, 0, MAP_WIDTH, MAP_HEIGHT);

        processKey(window);
        glfwSwapBuffers(window);
        glfwPollEvents();

        frames++;
        currentTime = std::chrono::system_clock::now();
        elapsed = currentTime - lastFrameOutput;
        if (elapsed.count() > 1.0) {
            std::cout << "Frame Rate: " << frames << "\n";
            lastFrameOutput = std::chrono::system_clock::now();
            frames = 0;
        }

        //std::this_thread::sleep_for(std::chrono::milliseconds(1));
    }

    glfwTerminate();
    return 0;
}

void processKey(GLFWwindow* window)
{
    if (glfwGetKey(window, GLFW_KEY_ESCAPE) == GLFW_PRESS) {
        for (int x = 50; x < 75; x++) {
            for (int y = 50; y < 75; y++) {
                glColor3f(0.0, 0.7, 0.7);
                glVertex2i(x, y);
            }
        }
    }
        
}



void errorCheck(int errorValue) {
    if (!(errorValue == 0)) {
        std::cout << "Error." << std::endl;
    }
}

void endProgram() {
    int a = 0;
    std::cout << "Press any key to continue." << "\n";
    std::cin >> a;
    exit(0);
}

